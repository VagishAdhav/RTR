#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    // function declaration 
    void PrintCUDADeviceProperties(void);

    // code
    PrintCUDADeviceProperties();
}

void PrintCUDADeviceProperties(void)
{
    // code
    printf("CUDA Information : \n");
    printf("===========================================================\n");

    hipError_t ret_cuda_rt;
    int dev_count;
    ret_cuda_rt = hipGetDeviceCount(&dev_count);
    if (ret_cuda_rt != hipSuccess)
    {
        printf("cuda Runtime API Error : %s\n", hipGetErrorString(ret_cuda_rt));
    }
    else if (dev_count == 0)
    {
        printf("No CUDA supported device on this system.\n");
        return;
    }
    else
    {
        int driverVersion = 0, runtimeVersion = 0;
        printf("Total number of CUDA supported GPU/Devices on this system : %d\n", dev_count);
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
        printf("**************CUDA DRIVER AND RUNTIME VERSION**************\n");
        printf("===========================================================\n");
        printf("CUDA Driver Version                                          : %d.%d\n", driverVersion/1000, (driverVersion % 100)/10);
        printf("CUDA Runtime Version                                         : %d.%d\n", runtimeVersion/1000, (runtimeVersion % 100)/10);
        for (int i = 0; i < dev_count; i++)
        {
            hipDeviceProp_t dev_prop;
            memset((void *)&dev_prop, 0, sizeof(hipDeviceProp_t));


            ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
            if (ret_cuda_rt != hipSuccess)
            {
                printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
                return;
            }
            printf("\n");
            printf("===========================================================\n");
            printf("**************GPU DEVICE GENERAL INFORMATION***************\n");
            printf("===========================================================\n");
            printf("GPU device number                                           :%d\n", i);
            printf("GPU device name                                             :%s\n", dev_prop.name);
            printf("GPU device compute capability                               :%d.%d\n", dev_prop.major, dev_prop.minor);
            printf("GPU devie clock rate                                        :%d\n", dev_prop.clockRate);
            printf("GPU device Type                                             :");
            if (dev_prop.integrated)
                printf("Integrated (On- Board)\n");
            else
                printf("Discrete (Card)\n");
            printf("\n");
            printf("===========================================================\n");
            printf("**************GPU DEVICE MEMORY INFORMATION****************\n");
            printf("===========================================================\n");
            printf("GPU device Total Memory                                     : %.0f GB, %.0f MB, %llu Bytes\n", ((float)dev_prop.totalGlobalMem/1048576.0f)/1024.0f, (float)dev_prop.totalGlobalMem/1048576.0f, (unsigned long long)dev_prop.totalGlobalMem);
            printf("GPU device Constant Memory                                  : %llu Bytes\n", (unsigned long long)dev_prop.totalConstMem);
            printf("GPU device Shared Memory per SMProcessor                    : %llu Bytes\n", (unsigned long long)dev_prop.sharedMemPerBlock);
            printf("\n");
            printf("===========================================================\n");
            printf("**************GPU DEVICE MULTIPROCESSOR INFORMATION********\n");
            printf("===========================================================\n");
            printf("GPU device number of SMProcessors                           :%lu\n", dev_prop.multiProcessorCount);
            printf("GPU device registers per SMProcessors                       :%lu\n", dev_prop.regsPerBlock);
            printf("\n");
            printf("===========================================================\n");
            printf("*******************GPU DEVICE THREAD INFORMATION***********\n");
            printf("===========================================================\n");
            printf("GPU Device maximum number of threads per SMProcessor       : %d\n", dev_prop.maxThreadsPerMultiProcessor);
            printf("GPU Device maximum number of threads per Block             : %d\n", dev_prop.maxThreadsPerBlock);
            printf("GPU Device Threads per Warp                                : %d\n", dev_prop.warpSize);
            printf("GPU Device maximum thread dimensions                       :(%d, %d, %d)\n", dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
            printf("GPU device maximum number of Grid Dimension                :(%d, %d, %d)\n", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);
            printf("\n");
            printf("===========================================================\n");
            printf("*******************GPU DEVICE DIVER INFORMATION************\n");
            printf("===========================================================\n");
            printf("GPU Device has ECC support                                  :%s\n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
            printf("GPU decvice CUDA driver mode (TCC or WDDM)                  :%s\n", dev_prop.tccDriver ? "TCC (Tesla compute cluster driver)" : "WDDM (Windows display driver module)");
#endif
            printf("************************************************************\n");

        }

    }
}
