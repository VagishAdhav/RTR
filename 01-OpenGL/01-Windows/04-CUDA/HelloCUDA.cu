// header file
// standard headers
#include <stdio.h>

// cuda headers
#include <hip/hip_runtime.h>

// global variables
const int iNumberOfArrayElements = 5;

float *hostInput1 = NULL;
float *hostInput2 = NULL;
float *hostOutput = NULL;

float *deviceInput1 = NULL;
float *deviceInput2 = NULL;
float *deviceOutput = NULL;

// CUDA Kernel
__global__ void vecAddGPU(float* in1, float* in2, float *out, int len)
{
    // code
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("vecAddGPU blockIdx.x %d, blockDim.x %d, threadIdx.x %d resultant index:%d\n",blockIdx.x,  blockDim.x, threadIdx.x, i);
    if (i < len)
    {
        out[i] = in1[i] + in2[i];
    }
}

// entry point function
int main(void)
{
    // function declaration
    void cleanup(void);

    // variable declaration
    int size = iNumberOfArrayElements * sizeof(float);
    hipError_t result = hipSuccess;

    // code
    // host memory allocation
    hostInput1 = (float *)malloc(size);
    if (hostInput1 == NULL)
    {
        printf("Host Memory allocation is failed for hostInput1 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostInput2 = (float *)malloc(size);
    if (hostInput2 == NULL)
    {
        printf("Host Memory allocation is failed for hostInput2 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    hostOutput = (float *)malloc(size);
    if (hostOutput == NULL)
    {
        printf("Host Memory allocation is failed for hostOutput array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // filling values into host arrays
    for (int index = 0; index < iNumberOfArrayElements; index++)
    {
        hostInput1[index] = 101.0 + (float)index;
        hostInput2[index] = 201.0 + (float)index;
    }

    // devide memory allocation
    result = hipMalloc((void **)&deviceInput1, size);
    if (result != hipSuccess)
    {
        printf("Device Memory allocation is failed for deviceInput1 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceInput2, size);
    if (result != hipSuccess)
    {
        printf("Device Memory allocation is failed for deviceInput2 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMalloc((void **)&deviceOutput, size);
    if (result != hipSuccess)
    {
        printf("Device Memory allocation is failed for deviceOutput array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // copy data from host memory into device memory
    result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Host to device data copy is failed for deviceInput1 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
        printf("Host to device data copy is failed for deviceInput2 array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid = dim3(iNumberOfArrayElements, 1, 1);
    dim3 dimBlock = dim3(1, 1, 1);

    // CUDA kernel for vector addition
    vecAddGPU <<< dimGrid, dimBlock >>> (deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements);

    // copy data from device array to hostarray
    result = hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
    {
        printf("Device to host data copy is failed for hostOutput array.\n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // vector addition on host
    for (int i = 0; i < iNumberOfArrayElements; i++)
    {
        printf("%f + %f = %f\n", hostInput1[i], hostInput2[i], hostOutput[i]);
    }

    // cleanup
    cleanup();

    return 0;

}

void cleanup(void)
{
    // code 
    if (deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = NULL;
    }

    if (deviceInput2)
    {
        hipFree(deviceInput2);
        deviceOutput = NULL;
    }

    if (deviceInput1)
    {
        hipFree(deviceInput1);
        deviceOutput = NULL;
    }

    if (hostOutput)
    {
        free(hostOutput);
        hostOutput = NULL;
    }

    if (hostInput2)
    {
        free(hostInput2);
        hostInput2 = NULL;
    }

    if (hostInput1)
    {
        free(hostInput1);
        hostInput1 = NULL;
    }

}